
#include <hip/hip_runtime.h>
#include <cstdio>


void inizializza(int i){

        hipDeviceProp_t prop;
        int count;

        hipGetDeviceCount (&count);

        hipGetDeviceProperties( &prop, i );
        printf("\n");
        printf("##############################################\n");
        printf("Device Name:\t\t %s\n",prop.name);
        printf("Shared Mem/block :\t %d\n",prop.sharedMemPerBlock);
        printf("Registri per blocco:\t %d\n",prop.regsPerBlock);
        printf("Warp size:\t\t %d\n",prop.warpSize);
        printf("Texature 1D :\t\t %d\n",prop.maxTexture1D);
        printf("MemPitch :\t\t %d\n",prop.memPitch);
        printf("##############################################\n");
        printf("\n");

}


