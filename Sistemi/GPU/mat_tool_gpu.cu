#include "hip/hip_runtime.h"
#define IDX(i,j,N) (i*N+j)
#include <cmath>
#include <cstdio>

#include "foo.h"


__global__ void pivoting(float *A, int k, int *pivrow, float *piv, int N){

	int it = threadIdx.x;
	int ib = blockIdx.x;

	int i = it + L_THREADS * ib;
	int passo = gridDim.x* L_THREADS;
		
	__shared__ float temp1[L_THREADS];
	__shared__ int temp2[L_THREADS];

	float temp;
	
	temp1[it] = 0.;
	temp2[it] = 0.;

	__syncthreads();
	
	//metto nella shared la matrice

	while(i<N){
		if(i>=k){
			temp = fabs(A[IDX(i,k,N)]);
			if(temp >= temp1[it]){
				temp1[it] = temp;
				temp2[it] = i;
			}
		}
		i +=passo;
	}
	__syncthreads();

		
	int l = blockDim.x / 2  ;
	
	//cerco il massimo 

	while(l != 0){
		if(it<l){
			if(temp1[it]<=temp1[it+l]){
				temp1[it] = temp1[it+l];
				temp2[it] = temp2[it+l];
			}
		}
		__syncthreads();
		l /= 2;
	}
	
	//il pivot sarà al posto zero di temp1
	if(it == 0){
		piv[ib] = temp1[0];
		pivrow[ib] = temp2[0];
	}

}

__global__ void pivoting3(float* A, int k, int* pivrow, float *piv, int N, const int N_BLOCKS){
	
		
	float temp=0;
	int temprow;
	for(int j=0; j < N_BLOCKS; j++){
		if(piv[j] >= temp ){
			temp = piv[j];
			temprow = pivrow[j];
		}
	}
	piv[0] = A[IDX(temprow,k,N)];
	pivrow[0] = temprow;
	return;
	
}


__global__ void scambio_riga(float* A, float* B, float* Ainv, int* pivrow, int k, int N){
	
	float temp1;
	float temp2;
	int j = threadIdx.x + blockDim.x*blockIdx.x;
	int passo = gridDim.x* blockDim.x;
	
	while(j < N){
		temp1 = A[IDX(pivrow[0],j,N)];
		A[IDX(pivrow[0],j,N)] = A[(IDX(k,j,N))];
		A[IDX(k,j,N)] = temp1;

		temp2 = Ainv[IDX(pivrow[0],j,N)];
		Ainv[IDX(pivrow[0],j,N)] = Ainv[IDX(k,j,N)];
		Ainv[IDX(k,j,N)] = temp2;

		j+=passo;
	}

	if(threadIdx.x + blockDim.x * blockIdx.x == 0){
		temp1 = B[pivrow[0]];
		B[pivrow[0]] = B[k];
		B[k] = temp1;
	}

	return;
}

__global__ void riscalamento(float* A, float* Ainv, float* B, int pivrow, int N, const float *piv){
	
	int j = threadIdx.x + blockDim.x*blockIdx.x;
	int passo = gridDim.x*blockDim.x;
	float pivot;
	pivot = 1/(*piv);

	while(j < N){
		A[IDX(pivrow,j,N)] *=pivot;
		Ainv[IDX(pivrow,j,N)] *= pivot;

		j += passo;
	}
	if(threadIdx.x + blockDim.x*blockIdx.x == 0)
		B[pivrow] *=pivot;
}

__global__ void Memorizzazione(float *A, float *app1, int N,int k){

	
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int passo = gridDim.x* blockDim.x;
	while(i < N){
		app1[i] = A[IDX(i,k,N)];
		i+=passo;
	}
	return;

}

__global__ void semplifico(float* A, float* Ainv, float* B, int pivrow, int N, float* app1 ){


	//cosa senza senso....
	int it = threadIdx.y;
	int jt = threadIdx.x;
	int jb = blockIdx.x;
	int ib = blockIdx.y;

	int i = it + THREADS*ib;
	int j = jt + THREADS*jb;
	//fine della cosa senza senso....

	if( i != pivrow ){
		A[IDX(i,j,N)] -=app1[i]*A[IDX(pivrow,j,N)];
		Ainv[IDX(i,j,N)] -= app1[i]*Ainv[IDX(pivrow,j,N)];
		if(j==0)
			B[i] -= app1[i]*B[pivrow];
	}
	return;


}
 
