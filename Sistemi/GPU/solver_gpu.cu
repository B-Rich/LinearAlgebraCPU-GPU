#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <iostream>

#include "mat_tool_gpu.h"
#include "foo.h"
#include "init.h"


#define H_D hipMemcpyHostToDevice 
#define D_H hipMemcpyDeviceToHost

#define CUDA_COPIA(a,b,c,d) hipMemcpy(a,b,c,d)



#define N_or 1024
//#define N_or 2048
//#define N_or 4096
//#define N_or 8192
//#define N_or 16384





#define IDX(i,j,N) (i*N+j)
#define DEV 0

#define imin(a,b) (a<b)?a:b


int main(){
	
	inizializza(DEV);


	hipSetDevice(DEV);

        int N = N_or;
        //const int N_BLOCKS = imin(32,(N+ L_THREADS-1)/L_THREADS );
	
	const int N_BLOCKS = N/16;	

	std::cout<<N_BLOCKS<<std::endl;

	float* A_h = new float[N*N];
	float* B_h = new float[N];
	float* Ainv_h = new float[N*N];
	

	
	float *A_dev, *B_dev, *Ainv_dev;

	//Matrice A
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++)
			A_h[IDX(i,j,N)] = 20*(drand48()-0.5);

	//Matrice Ainv= Id
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++){
			if(i == j) 
				Ainv_h[IDX(i,j,N)] = 1.;
			else
				Ainv_h[IDX(i,j,N)] = 0.;
		}

	//Termini Noti B
	for(int i=0; i<N; i++)
		B_h[i] = 20*(drand48()-0.5);
	

	hipEvent_t T1, T2;
	hipEventCreate(&T1);
	hipEventCreate(&T2);
	hipEventRecord(T1,0);

	//Allocazione di memoria sulla scheda
	hipMalloc((void**)&A_dev, N*N*sizeof(float));
	hipMalloc((void**)&B_dev, N*N*sizeof(float));
	hipMalloc((void**)&Ainv_dev, N*N*sizeof(float));
	


	//Trasferimento dati
	//hipMemcpy(A_dev, A_h, N*N*sizeof(float), hipMemcpyHostToDevice);

	CUDA_COPIA(A_dev,A_h,N*N*sizeof(float),H_D);
	hipMemcpy(Ainv_dev, Ainv_h, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_dev, B_h, N*sizeof(float), hipMemcpyHostToDevice);
	//dim3 threads(THREADS,THREADS);
	//dim3 blocks((N+THREADS-1)/THREADS,(N+THREADS-1)/THREADS);
	
	dim3 threads(THREADS,THREADS);
	dim3 blocks(N/THREADS,N/THREADS);


	int* pivrow_dev;
	float* piv_dev;
	float* app1_dev;
	
	hipMalloc((void**)&pivrow_dev,N_BLOCKS*sizeof(int));
	hipMalloc((void**)&piv_dev,N_BLOCKS*sizeof(float));
	hipMalloc((void**)&app1_dev,N*sizeof(float));
	
	for(int i=0; i<N; i++){
		/*
		pivoting<<<N_BLOCKS,L_THREADS>>>(A_dev, i, pivrow_dev, piv_dev, N);
		pivoting3<<<1,1>>>(A_dev, i, pivrow_dev, piv_dev, N, N_BLOCKS);
		scambio_riga<<<(N+L_THREADS-1)/L_THREADS, L_THREADS>>>(A_dev, B_dev, Ainv_dev, pivrow_dev, i, N);
		riscalamento<<<(N+L_THREADS-1)/L_THREADS, L_THREADS>>>(A_dev, Ainv_dev, B_dev, i, N, piv_dev);
		Memorizzazione<<<(N+L_THREADS-1)/L_THREADS,L_THREADS>>>(A_dev, app1_dev, N, i);
		semplifico<<<blocks,threads>>>(A_dev, Ainv_dev, B_dev, i, N, app1_dev);
		*/
	
		
		pivoting<<<N/L_THREADS,L_THREADS>>>(A_dev, i, pivrow_dev, piv_dev, N);
                pivoting3<<<1,1>>>(A_dev, i, pivrow_dev, piv_dev, N, N_BLOCKS);
                scambio_riga<<<N/L_THREADS, L_THREADS>>>(A_dev, B_dev, Ainv_dev, pivrow_dev, i, N);
                riscalamento<<<N/L_THREADS, L_THREADS>>>(A_dev, Ainv_dev, B_dev, i, N, piv_dev);
                Memorizzazione<<<N/L_THREADS,L_THREADS>>>(A_dev, app1_dev, N, i);
                semplifico<<<blocks,threads>>>(A_dev, Ainv_dev, B_dev, i, N, app1_dev);
	
	}
	
	hipFree(pivrow_dev);
	hipFree(piv_dev);
	hipFree(app1_dev);
	
	//A_dev --> Id
	//Ainv_dev --> inversa di A
	//B_dev --> soluzione
	

	hipMemcpy(A_h, A_dev, N*N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Ainv_h, Ainv_dev, N*N*sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(T2,0);
	hipEventSynchronize(T2);

	float diff_time;
	
	hipEventElapsedTime(&diff_time,T1,T2);

	printf("%i\t%f\n",N,diff_time/1000.);

/*	
	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++)
			printf("%f\t",A_h[j+i*N]);
		printf("\n");
	}
*/	

	hipEventDestroy(T1);
	hipEventDestroy(T2);
	hipFree(A_dev);
	hipFree(B_dev);
	hipFree(Ainv_dev);
	
	delete[](A_h);
	delete[](B_h);
	delete[](Ainv_h);
	
	return 0;
}
